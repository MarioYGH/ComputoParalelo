#include <iostream>
#include <fstream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <sstream>
#include <algorithm>

#define IDX(x, y, width) ((y)*(width) + (x))

// ------------------------
// Leer archivo PGM (P2)
bool loadPGM(const char* filename, float** outData, int* width, int* height) {
    std::ifstream file(filename);
    if (!file) return false;

    std::string line;
    std::getline(file, line);
    if (line != "P2") return false;

    while (std::getline(file, line)) {
        if (line.empty() || line[0] == '#') continue;

        std::istringstream dims(line);
        if (!(dims >> *width >> *height)) continue;
        break;
    }

    int maxVal = 255;
    while (std::getline(file, line)) {
        if (line.empty() || line[0] == '#') continue;

        std::istringstream mv(line);
        if (!(mv >> maxVal)) continue;
        break;
    }

    if (*width <= 0 || *height <= 0) return false;

    *outData = new float[(*width) * (*height)];
    int count = 0;
    while (std::getline(file, line)) {
        if (line.empty() || line[0] == '#') continue;

        std::istringstream ss(line);
        int val;
        while (ss >> val) {
            if (count < (*width) * (*height))
                (*outData)[count++] = (float)val;
        }
    }

    return count == (*width) * (*height);
}

// ------------------------
// Guardar archivo PGM
bool savePGM(const char* filename, float* data, int width, int height) {
    std::ofstream file(filename);
    if (!file) return false;

    file << "P2\n";
    file << width << " " << height << "\n";
    file << "255\n";

    for (int i = 0; i < width * height; ++i) {
        int val = std::min(255, std::max(0, (int)data[i]));
        file << val << ((i + 1) % width == 0 ? "\n" : " ");
    }

    return true;
}

// ------------------------
int nextPowerOfTwo(int x) {
    int power = 1;
    while (power < x) power *= 2;
    return power;
}

// ------------------------

void printElapsedTime(const char* label, float milliseconds) {
    std::cout << label << ": " << milliseconds << " ms\n";
}

void padImage(float*& data, int& width, int& height) {
    int newWidth = nextPowerOfTwo(width);
    int newHeight = nextPowerOfTwo(height);

    if (newWidth == width && newHeight == height) return;

    float* padded = new float[newWidth * newHeight]();

    for (int y = 0; y < height; ++y)
        for (int x = 0; x < width; ++x)
            padded[IDX(x, y, newWidth)] = data[IDX(x, y, width)];

    delete[] data;
    data = padded;
    width = newWidth;
    height = newHeight;

    std::cout << "Dimensiones ajustadas a: " << width << " x " << height << " (potencias de 2)\n";
}

void applyFFTShift(float* data, int width, int height) {
    int halfW = width / 2;
    int halfH = height / 2;

    // Swap top-left ↔ bottom-right
    for (int y = 0; y < halfH; ++y) {
        for (int x = 0; x < halfW; ++x) {
            int idx1 = y * width + x;
            int idx2 = (y + halfH) * width + (x + halfW);
            std::swap(data[idx1], data[idx2]);
        }
    }

    // Swap top-right ↔ bottom-left
    for (int y = 0; y < halfH; ++y) {
        for (int x = halfW; x < width; ++x) {
            int idx1 = y * width + x;
            int idx2 = (y + halfH) * width + (x - halfW);
            std::swap(data[idx1], data[idx2]);
        }
    }
}

int main() {
    const char* input = "barbara.ascii.pgm";
    const char* outputFFT = "resultadoFinalCuda.pgm";
    const char* outputReconstructed = "reconstruidaFinalCuda.pgm";

    float* h_input;
    int width, height;

    hipEvent_t startFFT, endFFT;
    hipEventCreate(&startFFT);
    hipEventCreate(&endFFT);

    if (!loadPGM(input, &h_input, &width, &height)) {
        std::cerr << "Error al cargar imagen.\n";
        return 1;
    }

    std::cout << "Dimensiones originales: " << width << " x " << height << std::endl;
    padImage(h_input, width, height);

    int size = width * height;

    hipfftReal* d_dataReal;
    hipfftComplex* d_dataComplex;

    hipMalloc(&d_dataReal, sizeof(hipfftReal) * size);
    hipMalloc(&d_dataComplex, sizeof(hipfftComplex) * (width * (height / 2 + 1)));

    hipMemcpy(d_dataReal, h_input, sizeof(float) * size, hipMemcpyHostToDevice);

    hipfftHandle planR2C;
    hipfftPlan2d(&planR2C, height, width, HIPFFT_R2C);

    hipEventRecord(startFFT);
    hipfftExecR2C(planR2C, d_dataReal, d_dataComplex);
    hipEventRecord(endFFT);
    hipEventSynchronize(endFFT);
    float fftTime = 0;
    hipEventElapsedTime(&fftTime, startFFT, endFFT);
    std::cout << "Tiempo de ejecución FFT: " << fftTime << " ms\n";

    hipfftComplex* h_complex = new hipfftComplex[width * (height / 2 + 1)];
    hipMemcpy(h_complex, d_dataComplex, sizeof(hipfftComplex) * width * (height / 2 + 1), hipMemcpyDeviceToHost);

    // ------------------------------------------
    // FIXED: Magnitude computation with symmetry
    float* h_magnitude = new float[width * height]();
    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            int fftX = x < (width / 2 + 1) ? x : width - x;

            if (fftX >= 0 && fftX < (width / 2 + 1)) {
                int idx = y * (width / 2 + 1) + fftX;
                float real = h_complex[idx].x;
                float imag = h_complex[idx].y;
                float mag = sqrtf(real * real + imag * imag);

                h_magnitude[IDX(x, y, width)] = logf(1.0f + mag);
            }
        }
    }

    // Normalize for PGM output
    float maxVal = *std::max_element(h_magnitude, h_magnitude + size);
    for (int i = 0; i < size; ++i)
        h_magnitude[i] = 255.0f * h_magnitude[i] / maxVal;

    applyFFTShift(h_magnitude, width, height);
    savePGM(outputFFT, h_magnitude, width, height);
    std::cout << "Imagen de la magnitud FFT guardada como " << outputFFT << "\n";

    // ------------------------------------------

    hipEvent_t startIFFT, endIFFT;
    hipEventCreate(&startIFFT);
    hipEventCreate(&endIFFT);
    // Inverse FFT (Reconstruction)
    hipfftHandle planC2R;
    hipfftPlan2d(&planC2R, height, width, HIPFFT_C2R);

    hipEventRecord(startIFFT);
    hipfftExecC2R(planC2R, d_dataComplex, d_dataReal);
    hipEventRecord(endIFFT);
    hipEventSynchronize(endIFFT);

    float ifftTime = 0;
    hipEventElapsedTime(&ifftTime, startIFFT, endIFFT);
    printElapsedTime("Tiempo de ejecución IFFT (reconstrucción)", ifftTime);

    float* h_reconstructed = new float[size];
    hipMemcpy(h_reconstructed, d_dataReal, sizeof(float) * size, hipMemcpyDeviceToHost);

    for (int i = 0; i < size; ++i)
        h_reconstructed[i] /= (width * height);

    savePGM(outputReconstructed, h_reconstructed, width, height);
    std::cout << "Imagen reconstruida guardada como " << outputReconstructed << "\n";

    // Cleanup
    delete[] h_input;
    delete[] h_complex;
    delete[] h_magnitude;
    delete[] h_reconstructed;
    hipFree(d_dataReal);
    hipFree(d_dataComplex);
    hipfftDestroy(planR2C);
    hipfftDestroy(planC2R);

    return 0;
}
